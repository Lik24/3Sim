#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from U.S. Dept. of Energy) All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or
 * without modification, are permitted provided that the
 * following conditions are met:
 *
 *     * Redistributions of source code must retain the above
 * copyright notice, this list of conditions and the following
 * disclaimer.
 *
 *     * Redistributions in binary form must reproduce the
 * above copyright notice, this list of conditions and the
 * following disclaimer in the documentation and/or other
 * materials provided with the distribution.
 *
 *     * Neither the name of the University of California,
 * Berkeley, nor the names of its contributors may be used to
 * endorse or promote products derived from this software
 * without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND
 * CONTRIBUTORS "AS IS" AND ANY EXVPRESS OR IMPLIED WARRANintES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANintES OF
 * MERCHANTABILITY AND FITNESS FOR A PARintCULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXVEMPLARY, OR CONSEQUENintAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSintTUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPintON) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 * Stefano Marchesini, Lawrence Berkeley National Laboratory, 2013
 */

#include <hip/hip_runtime.h>
#include <cusp/blas.h>
#include<cusp/csr_matrix.h>
#include<cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/copy.h>
#include <thrust/device_ptr.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"



template <typename IndexType>
        struct empty_row_functor
{
    typedef bool result_type;
    
    template <typename Tuple>
            __host__ __device__
            bool operator()(const Tuple& t) const
    {
        const IndexType a = thrust::get<0>(t);
        const IndexType b = thrust::get<1>(t);
        
        return a != b;
    }
};

/* Input Arguments */
#define	ROWPTR	prhs[0]
#define	NPTR    prhs[1]
#define	NNZ    prhs[2]

/* Output Arguments */
#define	ROW_OUT	plhs[0]


void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
    mxGPUArray  *Arow;
    mxGPUArray const *rowptr;
    mxInitGPU();     /* Initialize the MathWorks GPU API. */
    int nptr = lrint(mxGetScalar(NPTR));
    int nnz  = lrint(mxGetScalar(NNZ));
    const mwSize ndim= 1;
    const mwSize dimrow[]={mwSize(nnz)};
//      const mwSize dimptr[]={mwSize(nptr)};
//      mexPrintf("nrows=%d,nnz=%d\n", dimptr[0],dimcol[0]);
    
    // input output array
    rowptr = mxGPUCreateFromMxArray(ROWPTR);
    Arow  = mxGPUCreateGPUArray(ndim,dimrow,mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    int *d_rowptr =(int  *)(mxGPUGetDataReadOnly(rowptr));
    int *d_Arow =(int  *)(mxGPUGetData(Arow));
    
    // wrap with thrust::device_ptr
    thrust::device_ptr<int>    wd_Arow  (d_Arow);
    thrust::device_ptr<int>    wd_rowptr  (d_rowptr);

    /*-----------------------------------------------------------*/
    // ptr to row
    /*-----------------------------------------------------------*/
    
    thrust::fill(wd_Arow,wd_Arow+nptr, int(0));
    thrust::scatter_if( thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(nptr-1),
            wd_rowptr,
            thrust::make_transform_iterator(
            thrust::make_zip_iterator( thrust::make_tuple(wd_rowptr,wd_rowptr+1 ) ),
            empty_row_functor<int>()),
            wd_Arow);
    thrust::inclusive_scan(wd_Arow,wd_Arow+nnz, wd_Arow, thrust::maximum<int>());
    /*-----------------------------------------------------------*/
    
//bring back to matlab
    ROW_OUT = mxGPUCreateMxArrayOnGPU(Arow);
    //clean up
    mxGPUDestroyGPUArray(Arow);
    mxGPUDestroyGPUArray(rowptr);
    
    return;
}

