#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from U.S. Dept. of Energy) All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or
 * without modification, are permitted provided that the
 * following conditions are met:
 *
 *     * Redistributions of source code must retain the above
 * copyright notice, this list of conditions and the following
 * disclaimer.
 *
 *     * Redistributions in binary form must reproduce the
 * above copyright notice, this list of conditions and the
 * following disclaimer in the documentation and/or other
 * materials provided with the distribution.
 *
 *     * Neither the name of the University of California,
 * Berkeley, nor the names of its contributors may be used to
 * endorse or promote products derived from this software
 * without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND
 * CONTRIBUTORS "AS IS" AND ANY EXVPRESS OR IMPLIED WARRANTIES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
 * MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXVEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 * Stefano Marchesini, Lawrence Berkeley National Laboratory, 2013
 */

#include <hip/hip_runtime.h>
#include <cusp/complex.h>
#include <cusp/blas.h>
#include<cusp/csr_matrix.h>
#include<cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/copy.h>
#include <thrust/device_ptr.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"

/* Input Arguments */
#define	VAL	prhs[0]
#define	COL	prhs[1]
#define	ROWPTR	prhs[2]
// #define	NCOL    prhs[3]
// #define	NROW    prhs[4]
// #define	NNZ    prhs[5]
#define	XV    prhs[3]


/* Output Arguments */
#define	Y	plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
    
    mxGPUArray const *Aval;
    mxGPUArray const *Acol;
    mxGPUArray const *Aptr;
    mxGPUArray const *x;
    mxGPUArray  *y;
    
//     int nnzs = lrint(mxGetScalar(NCOL));
//     int nrows = lrint(mxGetScalar(NROW));
//     int nptr=nrows+1;
//     int nnz  = lrint(mxGetScalar(NNZ));
//     
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    
    /*get matlab variables*/
    Aval = mxGPUCreateFromMxArray(VAL);
    Acol = mxGPUCreateFromMxArray(COL);
    Aptr = mxGPUCreateFromMxArray(ROWPTR);
    x    = mxGPUCreateFromMxArray(XV);
    
    int nnz=mxGPUGetNumberOfElements(Acol);
    int nrowp1=mxGPUGetNumberOfElements(Aptr);
    int ncol =mxGPUGetNumberOfElements(x);

    
    mxComplexity isXVreal = mxGPUGetComplexity(x);
    mxComplexity isAreal = mxGPUGetComplexity(Aval);
    const mwSize ndim= 1;
    const mwSize dims[]={(mwSize) (nrowp1-1)};

    if (isAreal!=isXVreal)
    {
        mexErrMsgTxt("Aval and X must have the same complexity");
        return;
    }

    if(mxGPUGetClassID(Aval) != mxSINGLE_CLASS||
   mxGPUGetClassID(x)!= mxSINGLE_CLASS||
   mxGPUGetClassID(Aptr)!= mxINT32_CLASS||
   mxGPUGetClassID(Acol)!= mxINT32_CLASS){
     mexErrMsgTxt("usage: gspmv(single, int32, int32, single )");
     return;
    }
    
    //create output vector
    y = mxGPUCreateGPUArray(ndim,dims,mxGPUGetClassID(x),isAreal, MX_GPU_DO_NOT_INITIALIZE);
     
    
    /* wrap indices from matlab */
    typedef const int  TI;  /* the type for index */
    TI *d_col =(TI  *)(mxGPUGetDataReadOnly(Acol));
    TI *d_ptr =(TI  *)(mxGPUGetDataReadOnly(Aptr));
    // wrap with thrust::device_ptr
    thrust::device_ptr<TI>    wrap_d_col  (d_col);
    thrust::device_ptr<TI>    wrap_d_ptr  (d_ptr);
    // wrap with array1d_view 
    typedef typename cusp::array1d_view< thrust::device_ptr<TI> >   idx2Av;
    // wrap index arrays
    idx2Av colIndex (wrap_d_col , wrap_d_col + nnz);
    idx2Av ptrIndex (wrap_d_ptr , wrap_d_ptr + nrowp1);
           
    if (isAreal!=mxREAL){

        typedef const cusp::complex<float> TA;  /* the type for A */
        typedef const cusp::complex<float> TXV; /* the type for X */
        typedef cusp::complex<float> TYV; /* the type for Y */

        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + nnz);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + ncol);
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrowp1-1);
//        y2Av yIndex(wrap_d_y, wrap_d_y+ ncol);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrowp1-1, ncol, nnz, ptrIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
        
    }
     else{
         
        typedef const float TA;  /* the type for A */
        typedef const float TXV; /* the type for X */
        typedef float TYV; /* the type for Y */
   
        /* pointers from matlab */
        TA *d_val =(TA  *)(mxGPUGetDataReadOnly(Aval));
        TXV *d_x   =(TXV  *)(mxGPUGetDataReadOnly(x));
        TYV *d_y   =(TYV  *)(mxGPUGetData(y));
        
        // wrap with thrust::device_ptr!
        thrust::device_ptr<TA >    wrap_d_val  (d_val);
        thrust::device_ptr<TXV >    wrap_d_x    (d_x);
        thrust::device_ptr<TYV >    wrap_d_y  (d_y);
        // wrap with array1d_view 
        typedef typename cusp::array1d_view< thrust::device_ptr<TA > >   val2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TXV > >   x2Av;
        typedef typename cusp::array1d_view< thrust::device_ptr<TYV > >   y2Av;
        
        // wrap  arrays
        val2Av valIndex (wrap_d_val , wrap_d_val + nnz);
        x2Av xIndex   (wrap_d_x   , wrap_d_x   + ncol);
        //y2Av yIndex(wrap_d_y, wrap_d_y+ ncol);        
        y2Av yIndex(wrap_d_y, wrap_d_y+ nrowp1-1);
        
        // combine info in CSR matrix
        typedef  cusp::csr_matrix_view<idx2Av,idx2Av,val2Av> DeviceView;
        
        DeviceView As(nrowp1-1, ncol, nnz, ptrIndex, colIndex, valIndex);
                
        // multiply matrix
        cusp::multiply(As, xIndex, yIndex);
        
    }

    Y = mxGPUCreateMxArrayOnGPU(y);
    
    mxGPUDestroyGPUArray(Aval);
    mxGPUDestroyGPUArray(Aptr);
    mxGPUDestroyGPUArray(Acol);
    mxGPUDestroyGPUArray(x);
    mxGPUDestroyGPUArray(y);

    return;
}

