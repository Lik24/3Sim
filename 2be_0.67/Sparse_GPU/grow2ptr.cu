#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from U.S. Dept. of Energy) All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or
 * without modification, are permitted provided that the
 * following conditions are met:
 *
 *     * Redistributions of source code must retain the above
 * copyright notice, this list of conditions and the following
 * disclaimer.
 *
 *     * Redistributions in binary form must reproduce the
 * above copyright notice, this list of conditions and the
 * following disclaimer in the documentation and/or other
 * materials provided with the distribution.
 *
 *     * Neither the name of the University of California,
 * Berkeley, nor the names of its contributors may be used to
 * endorse or promote products derived from this software
 * without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND
 * CONTRIBUTORS "AS IS" AND ANY EXVPRESS OR IMPLIED WARRANintES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANintES OF
 * MERCHANTABILITY AND FITNESS FOR A PARintCULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXVEMPLARY, OR CONSEQUENintAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSintTUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPintON) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 * Stefano Marchesini, Lawrence Berkeley National Laboratory, 2013
 */

#include <hip/hip_runtime.h>
#include <cusp/blas.h>
#include<cusp/csr_matrix.h>
#include<cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/copy.h>
#include <thrust/device_ptr.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"


/* Input Arguments */
#define	ROW	prhs[0]
#define	NPTR    prhs[1]
#define	NNZ    prhs[2]

/* Output Arguments */
#define	ROW_OUT	plhs[0]




void mexFunction(int nlhs, mxArray * plhs[], int nrhs,const mxArray * prhs[]){
    mxGPUArray const *Arow;
    mxGPUArray  *rowptr;
    mxInitGPU();     /* Initialize the MathWorks GPU API. */
   int nptr = lrint(mxGetScalar(NPTR));
    int nnz  = lrint(mxGetScalar(NNZ));
    const mwSize ndim= 1;    
//    const mwSize dimcol[]={mwSize(nnz)};    
     const mwSize dimptr[]={mwSize(nptr)};    
//      mexPrintf("nrows=%d,nnz=%d\n", dimptr[0],dimcol[0]);

    // input output array 
    Arow = mxGPUCreateFromMxArray(ROW);
      rowptr = mxGPUCreateGPUArray(ndim,dimptr,mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
//          mexPrintf("created matrix, nrows=%d,nnz=%d\n", nptr,nnz);
    // pointer from matlab
    int *d_Arow =(int  *)(mxGPUGetDataReadOnly(Arow));    
    int *d_rowptr =(int  *)(mxGPUGetData(rowptr));

    // wrap with thrust::device_ptr
    thrust::device_ptr<int>    wrap_d_Arow  (d_Arow);
    thrust::device_ptr<int>    wrap_d_rowptr  (d_rowptr);
    // convert to ptr 
      thrust::lower_bound(wrap_d_Arow,
                        wrap_d_Arow+nnz,
                        thrust::counting_iterator<int>(0),
                        thrust::counting_iterator<int>(nptr),
                        wrap_d_rowptr);
    //bring back to matlab
    ROW_OUT = mxGPUCreateMxArrayOnGPU(rowptr);
    //clean up
    mxGPUDestroyGPUArray(Arow);
    mxGPUDestroyGPUArray(rowptr);

    return;
}

